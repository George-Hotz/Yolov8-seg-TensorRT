#include "hip/hip_runtime.h"
#include "preprocess.h"
#include "yolo_infer.h"

// memory_ctrl::Memory<uint8_t> preprocess_buffer; //预处理相关的内存申请
// AffineMatrix affine_matrixs;             //仿射变换矩阵

//对齐操作，对齐数：align
inline int upbound(int n, int align = 32) { return (n + align - 1) / align * align; } 

__global__ void warpaffine_kernel(
    uint8_t *src, int src_line_size, int src_width, int src_height, 
    float *dst, int dst_width, int dst_height, uint8_t const_value_st,
    float * warp_affine_matrix_2_3)
{
  float mean[] {0.485, 0.456, 0.406};
  float std[] {0.229, 0.224, 0.225};

  int dx = blockDim.x * blockIdx.x + threadIdx.x;
  int dy = blockDim.y * blockIdx.y + threadIdx.y;

  //dst_width  网络输入宽度
  //dst_height 网络输入高度
  if (dx >= dst_width || dy >= dst_height) return;

  float m_x1 = warp_affine_matrix_2_3[0];  //d2i
  float m_y1 = warp_affine_matrix_2_3[1];
  float m_z1 = warp_affine_matrix_2_3[2];
  float m_x2 = warp_affine_matrix_2_3[3];
  float m_y2 = warp_affine_matrix_2_3[4];
  float m_z2 = warp_affine_matrix_2_3[5];

  float src_x = m_x1 * dx + m_y1 * dy + m_z1;
  float src_y = m_x2 * dx + m_y2 * dy + m_z2;
  float c0, c1, c2;

  //src_width  原图宽度
  //src_height 原图高度
  if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height) {
    // out of range
    c0 = const_value_st;
    c1 = const_value_st;
    c2 = const_value_st;
  } else {
    int y_low = floorf(src_y);
    int x_low = floorf(src_x);
    int y_high = y_low + 1;
    int x_high = x_low + 1;

    uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
    float ly = src_y - y_low;
    float lx = src_x - x_low;
    float hy = 1 - ly;
    float hx = 1 - lx;
    float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
    uint8_t *v1 = const_value;
    uint8_t *v2 = const_value;
    uint8_t *v3 = const_value;
    uint8_t *v4 = const_value;
    if (y_low >= 0) {
      if (x_low >= 0) 
        v1 = src + y_low * src_line_size + x_low * 3;

      if (x_high < src_width) 
        v2 = src + y_low * src_line_size + x_high * 3;
    }

    if (y_high < src_height) {
      if (x_low >= 0) 
        v3 = src + y_high * src_line_size + x_low * 3;

      if (x_high < src_width) 
        v4 = src + y_high * src_line_size + x_high * 3;
    }

    // same to opencv
    c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
    c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
    c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
  }

  float t = c2;
  c2 = c0;
  c0 = t;

  // normalization
  c0 = c0 / 255.0f;
  c1 = c1 / 255.0f;
  c2 = c2 / 255.0f;

  //imagenet normalization
  // c0 = (c0-mean[0])/std[0];
  // c1 = (c1-mean[1])/std[1];
  // c2 = (c2-mean[2])/std[2];

  // rgbrgbrgb to rrrgggbbb
  int area = dst_width * dst_height;
  float *pdst_c0 = dst + dy * dst_width + dx;
  float *pdst_c1 = pdst_c0 + area;
  float *pdst_c2 = pdst_c1 + area;
  *pdst_c0 = c0;
  *pdst_c1 = c1;
  *pdst_c2 = c2;

}


__global__ void bgrToGray_Brightness_cuda(const uint8_t* image, int width, int height, float* brightness) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = (y * width + x) * 3;  // 每个像素有3个通道 (R, G, B)
        
        uint8_t gray = static_cast<uint8_t>(0.299f * image[index] + 0.587f * image[index + 1] + 0.114f * image[index + 2]);

        atomicAdd(brightness, static_cast<float>(gray));
    }
}



/// @brief 判断输入图像的亮度低于阈值
/// @param input 输入图像
/// @param threshold 图像亮度阈值(0~255)
/// @return 低于阈值返回true，否则返回false
bool check_Brightness(const cv::Mat &input, const uint8_t threshold)
{
  //输入图像是uint8_t类型
  size_t input_size = input.rows * input.cols * 3;
  uint8_t *cache_input_gpu = nullptr;
  float *brightness_gpu;
  float brightness_cpu;

  hipStream_t stream_ = nullptr;  
  hipStreamCreate(&stream_);
  CUDA_CHECK(hipMalloc((void**)&brightness_gpu, sizeof(float)));
  CUDA_CHECK(hipMalloc((void**)&cache_input_gpu, input_size));
  CUDA_CHECK(hipMemset(brightness_gpu, 0, sizeof(float))); //初始化为0

  //CUDA_CHECK(hipMemcpy(cache_input_gpu, (uint8_t *)input.ptr(), input_size, hipMemcpyHostToDevice));

  CUDA_CHECK(hipMemcpyAsync(cache_input_gpu, (uint8_t *)input.ptr(), input_size, 
                             hipMemcpyHostToDevice, stream_));

  // 定义块和网格维度
  dim3 blockDim(32, 32);
  dim3 gridDim((input.cols + blockDim.x - 1) / blockDim.x, (input.rows + blockDim.y - 1) / blockDim.y);

  // 启动内核
  bgrToGray_Brightness_cuda<<<gridDim, blockDim, 0, stream_>>>(cache_input_gpu, 
                                                               input.cols, 
                                                               input.rows, 
                                                               brightness_gpu);

  // 将结果复制回主机
  CUDA_CHECK(hipMemcpyAsync(&brightness_cpu, brightness_gpu, sizeof(float), 
                             hipMemcpyDeviceToHost, stream_));

  //CUDA流同步
  CUDA_CHECK(hipStreamSynchronize(stream_)); 

  // 释放设备内存
  CUDA_CHECK(hipFree(cache_input_gpu));
  CUDA_CHECK(hipFree(brightness_gpu));

  brightness_cpu /= (input.cols * input.rows);

  auto result = brightness_cpu < threshold;
  std::cout<< "  平均亮度: "<< brightness_cpu
           << "，阈值亮度: "<< (float)threshold
           << "，结果: "<< result << std::endl;

  if(result)
    return true;  //亮度过低，需要补光增强
  else
    return false; //亮度正常，无需补光增强

}


void cuda_preprocess(
    uint8_t *src, int src_width, int src_height,
    float *dst, int dst_width, int dst_height,
    memory_ctrl::Memory<uint8_t> &preprocess_buffer,
    AffineMatrix &affine_matrixs)
{

  // 计算变换矩阵
  affine_matrixs.compute(std::make_tuple(src_width, src_height),  //from(原图)
                         std::make_tuple(dst_width, dst_height)); //to(640,640)

  size_t size_matrix = upbound(sizeof(affine_matrixs.d2i), 32);
  size_t size_image = src_width * src_height * 3;

  //GPU申请内存
  uint8_t *gpu_workspace = preprocess_buffer.gpu(size_matrix + size_image);
  float *affine_matrix_device = (float *)gpu_workspace;
  uint8_t *img_buffer_device = gpu_workspace + size_matrix;

  //CPU申请内存
  uint8_t *cpu_workspace = preprocess_buffer.cpu(size_matrix + size_image);
  float *affine_matrix_host = (float *)cpu_workspace;
  uint8_t *img_buffer_host = cpu_workspace + size_matrix;

  //对申请的CPU内存赋值
  memcpy(affine_matrix_host, affine_matrixs.d2i, size_matrix);//赋值affine_matrixs的数据
  memcpy(img_buffer_host, src, size_image);  //赋值src的数据

  hipStream_t stream_ = nullptr;  //加速
  hipStreamCreate(&stream_);
  // CPU-->GPU转移内存
  CUDA_CHECK(hipMemcpyAsync(img_buffer_device, img_buffer_host, size_image, 
                                hipMemcpyHostToDevice, stream_));
  CUDA_CHECK(hipMemcpyAsync(affine_matrix_device, affine_matrix_host, size_matrix,
                                hipMemcpyHostToDevice, stream_));

  //CUDA流同步
  CUDA_CHECK(hipStreamSynchronize(stream_)); 

  dim3 grid((dst_width + 31) / 32, (dst_height + 31) / 32);
  dim3 block(32, 32);

  // 调用kernel函数
  warpaffine_kernel<<<grid, block, 0, stream_>>>(
      img_buffer_device, src_width * 3, src_width,
      src_height, dst, dst_width, dst_height, 0, 
      affine_matrix_device);
}


// 使用cuda预处理所有步骤
void Preprocess_gpu(const cv::Mat &src, int inputW, int inputH, float *input_device_buffer,
                    memory_ctrl::Memory<uint8_t> &preprocess_buffer, AffineMatrix &affine_matrixs)
{
  cuda_preprocess((uint8_t *)src.ptr(), src.cols, src.rows, input_device_buffer, inputW, inputH,
                   preprocess_buffer, affine_matrixs);
}


////////////////////////////////  CPU Part  ////////////////////////////////

void Preprocess_cpu(const cv::Mat &img, int inputW, int inputH, float *input_host_buffer)
{

    cv::Mat resized;
    cv::resize(img, resized, cv::Size(inputW, inputH));
    cv::Mat rgb;
    cv::cvtColor(resized, rgb, cv::COLOR_BGR2RGB);
    cv::Mat normalized;
    rgb.convertTo(normalized, CV_32FC3);
    cv::subtract(normalized, cv::Scalar(127.5, 127.5, 127.5), normalized);
    cv::divide(normalized, cv::Scalar(127.5, 127.5, 127.5), normalized);
    // split it into three channels
    std::vector<cv::Mat> nchw_channels;
    cv::split(normalized, nchw_channels);

    for (auto &img : nchw_channels)
    {
        img = img.reshape(1, 1);
    }

    cv::Mat nchw;
    cv::hconcat(nchw_channels, nchw);

    memcpy(input_host_buffer, nchw.data, 3 * inputH * inputW * sizeof(float));
}
